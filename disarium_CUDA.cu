
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <assert.h>
#include <math.h> 

using namespace std;

const int SIZE = 1000000;

int *generate_input();
__global__ void _cuda_parallel_Disarium(int count, int* test_data, int* output);
__device__ bool _cuda_parallel_isDisariumNumber(int number);
__device__ int _cuda_parallel_countDigits(int number);
bool isDisariumNumber(int number);
int countDigits(int number);
void method(int* input, int threads_number, int blocks_number);

int main() {
    int threads_number_const = 1;
    int blocks_number = ceil(SIZE/threads_number_const) + 1;
    int* input = generate_input();
    int* output_sequential = (int *)malloc(sizeof(int) * SIZE);

    for(int i = 0; i < SIZE; i++){
        output_sequential[i]=isDisariumNumber(input[i]);
    }

    printf("Meastrements with changing threads number (1-30): \n");
    for(int i = threads_number_const; i < 31; i+=1){
        int blocks_number = ceil(SIZE/i) + 1;
        method(input, i, blocks_number);
    }
}

void method(int* input, int threads_number, int blocks_number){
    int* output_parallel = (int *)malloc(sizeof(int) * SIZE);
        
    // Copy data to device
    int* d_input;
    hipMalloc(&d_input, SIZE * sizeof(int));
    int* d_output;
    hipMalloc(&d_output, SIZE * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_input, input, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, output_parallel, SIZE * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start);
    // Launch kernel
    _cuda_parallel_Disarium<<<blocks_number, threads_number>>>(SIZE, d_input, d_output);
    hipEventRecord(stop);
        
    // Copy results back to device
    hipDeviceSynchronize();
    hipMemcpy(output_parallel, d_output, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipFree(d_input);
    hipFree(d_output);
    printf("time for %d threads: %lf ms\n", threads_number, milliseconds);
}

int *generate_input(){
    int *input = (int *)malloc(sizeof(int)* SIZE);
    assert(input != NULL);
    int m;
    for(m=0; m<SIZE; m++){
        int number = m+1;
        input[m]=number;
    }
    return input;
}

__global__ void _cuda_parallel_Disarium(int size, int* input, int* output) {

     int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

     while (globalIdx < size) {
        output[globalIdx] = _cuda_parallel_isDisariumNumber(input[globalIdx]);

        globalIdx +=  blockDim.x * gridDim.x;
        __syncthreads();
     }
}

__device__ bool _cuda_parallel_isDisariumNumber(int number){
    int count_digits = _cuda_parallel_countDigits(number);
    int sum = 0;
    int x = number;
    while (x)
    {
        int r = x%10;
        sum = sum + pow(r, count_digits--);
        x = x/10;
    }
    return (sum==number);
}

__device__ int _cuda_parallel_countDigits(int number)
{
    int count_digits = 0;
    int x = number;

    while (x)
    {
        x = x/10;
        count_digits++;
    }
    return count_digits;
}

bool isDisariumNumber(int number){
    int count_digits = countDigits(number);
    int sum = 0;
    int x = number;
    while (x)
    {
        int r = x%10;
        sum = sum + pow(r, count_digits--);
        x = x/10;
    }
    return sum==number;
}

int countDigits(int number)
{
    int count_digits = 0;
    int x = number;

    while (x)
    {
        x = x/10;
        count_digits++;
    }
    return count_digits;
}